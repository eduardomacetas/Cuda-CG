
#include <hip/hip_runtime.h>
#include <stdio.h>

//#define N 50
//#define THREADS_PER_BLOCK 10
//#define RADIUS 3
//#define BLOCK_SIZE 20

#define N 1500
//#define THREADS_PER_BLOCK 10
#define RADIUS 3
#define BLOCK_SIZE 3

__global__ void stencil_ld(int *in, int *out) {
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    // Leer elementos de entrada en la memoria compartida
    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) {
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    __syncthreads();

    // Aplicamos el stencil:
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++)
        result += temp[lindex + offset];

    // Almacena el resultado:
    out[gindex] = result;
}

void random_ints(int *V, int n){
    int i;
    for (i = 0; i < n; i++)
        V[i] = rand() % 5;
}

void print_vect(int *V, int n){
    int i;
    for (i = 0; i < n; i++)
		printf("%d\t", V[i]);
    printf("\n");
}

int main(void){
    int *a, *b;
    int *d_a, *d_b;
    int size = N * sizeof(int);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    random_ints(a, N);

    // Crear los eventos:
    hipEvent_t start, stop;
    hipEventCreate(&start); 
    hipEventCreate(&stop);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    // Registrar eventos alrededor del lanzamiento del kernel
    hipEventRecord(start); // Donde 0 es la secuencia predeterminada
    stencil_ld<<<(N + BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_a, d_b);
    
    hipEventRecord(stop);

    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    
    // Para calcular el tiempo:
    float time = 0;
    hipEventElapsedTime(&time, start, stop);

    print_vect(a, N);
    printf("\n");
    print_vect(b, N);
    printf("\n %fn milisegundos \n", time); // Imprimir el tiempo

    free(a);
    free(b);

    hipFree(d_a);
    hipFree(d_b);
    return 0;
}

// nvcc vect_stencil.cu -o v
// ./v