
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
using namespace std;


#define N (3*3)
#define THREADS_PER_BLOCK 3

void random_ints(int* a, int M)
{
   int i;
   for (i = 0; i < M; ++i)
        a[i] = rand()%5;
}

__global__ void multi(int *a, int *b, int *c,int n) {
    int suma = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y ; 
    int col = blockIdx.x * blockDim.x + threadIdx.x ;

    if (row <n && col<n){
        for(int i=0;i<N;++i){
        suma+= a[row*n+i] * b[i*n+col];
        }
    }
    c[row*n+col] = suma;
}

void imprimir(int *a){
	for(int i=0;i<N;i++)
		printf ("%d ",a[i]);
		printf("\n");
	printf("\n");
}

int main(void){
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; //device copies of a,b,c
	int size = N*sizeof(int);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // add<<<1,N>>>(d_a, d_b, d_c);
	// cladd<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
	multi<<<(N + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c,N);
	
	
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	imprimir(a);
	imprimir(b);
	imprimir(c);

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
