
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <math.h>
using namespace std;

#define filas 4
#define columnas 4
#define threadsPB 16

void randonInt(double **& matrix)
{
    for(int i=0;i<filas;++i){
	for(int j=0;j<columnas;++j)
            matrix[i][j]=rand() % 10 + 1;
    }
}

void createMatrixHostCUDA(double**& host, double**& device, double **& aux, int tamano, int f, int c ){
    host = (double **)malloc(f*sizeof(double*));
    host[0] = (double *)malloc(tamano);

    aux = (double **)malloc(f*sizeof(double*));
    hipMalloc((void **) &aux[0], tamano);
    hipMalloc((void **) &device, f*sizeof(double*));
    //cudaMalloc((void **)&(device[0]),size);

    for (int i=1; i<f;++i){
        host[i]=host[i-1]+c;
        aux[i]=aux[i-1]+c;
    }
    hipMemcpy(device, aux, f*sizeof(double*), hipMemcpyHostToDevice);
}

__global__ void sigmoid(double ** A,double ** C) {
     
     //printf("asdfas ");

     int i= threadIdx.x  + blockIdx.x * blockDim.x;
     int j = threadIdx.y + blockIdx.y * blockDim.y;

     if (i <filas && j <columnas){
         C[i][j]= 1.0/(1+exp(-A[i][j]));

//       printf("i: %i\t j: %i\n" ,i,j );
//       printf("2: %f \n" ,A[i][j] );
     }
}

__global__ void sigmoidGradient(double ** A, double ** C) {
     
     //printf("asdfas ");

     int i= threadIdx.x  + blockIdx.x * blockDim.x;
     int j = threadIdx.y + blockIdx.y * blockDim.y;

     if (i <filas && j <columnas){
         C[i][j]= (1.0/(1+exp(-A[i][j])))*(1-(1.0/(1+exp(-A[i][j]))));
     }
}

__global__ void scalarMult(double ** A, double s,double ** C) {
     
     //printf("asdfas ");

     int i= threadIdx.x  + blockIdx.x * blockDim.x;
     int j = threadIdx.y + blockIdx.y * blockDim.y;

     if (i <filas && j <columnas){
	 C[i][j]= A[i][j]* s;
//       printf("i: %i\t j: %i\n" ,i,j );
//       printf("2: %f \n" ,A[i][j] );
     }
}

void print(double ** a){
	for(int i=0;i<filas;++i){
	    for(int j=0;j<columnas;++j){
            cout<<a[i][j]<<'\t';
        }
	cout<<endl;
    }
	cout<<endl;
}


int main()
{
	//srand (time(NULL));
	double **a, **c;
	double **d_a, **d_c;
	double **a_aux, **c_aux;
	int size = filas * columnas * sizeof(double*);

    // CreateMatrix => host, device, aux, size, f, c
	createMatrixHostCUDA(a,d_a,a_aux,size,filas,columnas);
	createMatrixHostCUDA(c,d_c,c_aux,size,filas,columnas);

    // Genero mi matriz de randoms 
    // 4 filas y 4 columnas
	randonInt(a);
	
	hipMemcpy(a_aux[0], a[0], size, hipMemcpyHostToDevice);

	dim3 threadPerBlock(threadsPB, threadsPB);
	dim3 blockPerGrid((filas+threadPerBlock.x-1)/threadPerBlock.x,(columnas+threadPerBlock.y-1)/threadPerBlock.y);
    scalarMult<<<blockPerGrid,threadPerBlock>>>(d_a,2,d_c);

	hipMemcpy(c[0],c_aux[0], size, hipMemcpyDeviceToHost);
	
	hipFree(d_a);hipFree(d_c);
	hipFree(a_aux[0]);hipFree(c_aux[0]);

    cout<<"=============="<<endl;
    cout<<"Matriz A: "<<endl;
    cout<<"=============="<<endl;
	print (a);
 
	cout<<"=============="<<endl;
    cout<<"Matriz C: "<<endl;
    cout<<"=============="<<endl;;
	print(c);

	free(a); free(c);
	return 0;
}


