
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//#define N 10

#define N (4*4)
#define THREADS_PER_BLOCK 5	

#define g 10/2
void random_ints(int* a, int M)
{
   int i;
   for (i = 0; i < M; ++i)
        a[i] = rand()/10000000;
}

__global__ void add(int *a, int *b, int *c) {
	//c[threadIdx.x]=	a[threadIdx.x]+ b[threadIdx.x];

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];


	
	
}

void imprimir(int *a){
	for(int i=0;i<N;i++)
		printf ("%d ",a[i]);
		printf("\n");
	printf("\n");
}

int main(void){
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; //device copies of a,b,c
	int size = N*sizeof(int);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

//	add<<<1,N>>>(d_a, d_b, d_c);
	add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
	
	
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	imprimir(a);
	imprimir(b);
	imprimir(c);

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}

// nvcc suma.cu -o v
// ./v

